#include "hip/hip_runtime.h"
/*
    bwa_gpu a set of tools which allow short sequence alignment using the Burrows-Wheeler
    transform usign both CPU and GPU approaches.
    Copyright (C) 2011  Jose Salavert Torres, Ignacio Blanquer Espert,
                        Andres Tomas Dominguez, Vicente Hernandez Garcia,
	 		Ignacio Medina Castello, Joaquin Tarraga Gimenez,
			Joaquin Dopazo Blazquez

    Contact e-mail: josator@fiv.upv.es, iblanque@dsic.upv.es, atomas@dsic.upv.es,
                    vhernand@dsic.upv.es, imedina@cipf.es, jtarraga@cipf.es,
                    jdopazo@cipf.es

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program. If not, see <http://www.gnu.org/licenses/>.
*/

#include"BW_io.h"
#include"BW_gpu.cuh"

#include <hip/hip_runtime.h>
//#include "cuPrintf.cu"

#if defined VECTOR_O_32BIT_COMPRESSION

__device__ unsigned int getOcompValueGPU(size_t n, size_t m, comp_matrix O) {

  size_t pos, desp;

  pos  = m / 32;
  desp = m % 32;

  return O.desp[n][pos] + __popc( O.count[n][pos] << (32 - (desp + 1)) );

}

#elif defined VECTOR_O_64BIT_COMPRESSION

__device__ unsigned int getOcompValueGPU(size_t n, size_t m, comp_matrix O) {

  size_t pos, desp;

  pos  = m / 64;
  desp = m % 64;

  return O.desp[n][pos] + __popcll( O.count[n][pos] << (64 - (desp + 1)) );

}

#endif

void readCompMatrixGPU(comp_matrix *matrix, const char *directory, const char *name) {

  size_t err=0;
  hipError_t error;
  FILE *fp;

  char path[500];

  path[0]='\0';
  strcat(path, directory);
  strcat(path, "/");
  strcat(path, name);
  strcat(path, ".desp");

  fp  = fopen(path,  "rb+");
  checkFileOpen(fp, path);

  err = fread(&matrix->siz,    sizeof(size_t),  1, fp);
  checkFileRead(err, 1, path);

  err = fread(&matrix->n_desp, sizeof(size_t),  1, fp);
  checkFileRead(err, 1, path);

  err = fread(&matrix->m_desp, sizeof(size_t),  1, fp);
  checkFileRead(err, 1, path);

  for (size_t i=0; i<matrix->n_desp; i++) {
    hipHostMalloc((void**) &matrix->desp[i], matrix->m_desp * sizeof(unsigned int));
    manageCudaError();
    err = fread(matrix->desp[i], sizeof(unsigned int), matrix->m_desp, fp);
    checkFileRead(err, matrix->m_desp, path);
  }

  fclose(fp);

#if defined VECTOR_O_32BIT_COMPRESSION

  path[0]='\0';
  strcat(path, directory);
  strcat(path, "/");
  strcat(path, name);
  strcat(path, ".count");

  fp  = fopen(path,  "rb+");
  checkFileOpen(fp, path);

  err = fread(&matrix->n_count,   sizeof(size_t),  1, fp);
  checkFileRead(err, 1, path);

  err = fread(&matrix->m_count,   sizeof(size_t),  1, fp);
  checkFileRead(err, 1, path);

  for (size_t i=0; i<matrix->n_count; i++){
    hipHostMalloc((void**) &matrix->count[i], matrix->m_count * sizeof(unsigned int));
    manageCudaError();
    err = fread(matrix->count[i], sizeof(unsigned int), matrix->m_count, fp);
    checkFileRead(err, matrix->m_count, path);
  }

  fclose(fp);

#elif defined VECTOR_O_64BIT_COMPRESSION

  path[0]='\0';
  strcat(path, directory);
  strcat(path, "/");
  strcat(path, name);
  strcat(path, ".count");

  fp  = fopen(path,  "rb+");
  checkFileOpen(fp, path);

  err = fread(&matrix->n_count, sizeof(size_t),  1, fp);
  checkFileRead(err, 1, path);

  err = fread(&matrix->m_count, sizeof(size_t),  1, fp);
  checkFileRead(err, 1, path);

  for (size_t i=0; i<matrix->n_count; i++){

    hipHostMalloc((void**) &matrix->count[i], matrix->m_count * sizeof(unsigned long long));
    manageCudaError();
    err = fread(matrix->count[i], sizeof(unsigned long long), matrix->m_count, fp);
    checkFileRead(err, matrix->m_count, path);

  }

  fclose(fp);

#endif

}

void copyVectorGPU(vector *device, vector *host, size_t data_size) {

  hipError_t error;

  device->n = host->n;
  //  printf("device->n = %d\n", device->n);
  hipMalloc((void**) &device->vector,  device->n * data_size);
  //  printf("hipMalloc done !!\n");
  manageCudaError();
  hipMemcpy(device->vector, host->vector, device->n * data_size, hipMemcpyHostToDevice);
  //  printf("hipMemcpy done !!\n");
  manageCudaError();

}

void copyCompMatrixGPU(comp_matrix *device, comp_matrix *host) {

  hipError_t error;

  device->siz    = host->siz;
  device->n_desp = host->n_desp;
  device->m_desp = host->m_desp;

  for (size_t i=0; i<device->n_desp; i++) {
    hipMalloc((void**) &device->desp[i], device->m_desp * sizeof(unsigned int));
    manageCudaError();
    hipMemcpy(device->desp[i], host->desp[i], host->m_desp * sizeof(unsigned int), hipMemcpyHostToDevice);
    manageCudaError();
  }

#if defined   VECTOR_O_32BIT_COMPRESSION
  
  device->n_count = host->n_count;
  device->m_count = host->m_count;

  for (size_t i=0; i<device->n_count; i++) {
    hipMalloc((void**) &device->count[i], device->m_count * sizeof(unsigned int));
    manageCudaError();
    hipMemcpy(device->count[i], host->count[i], host->m_count * sizeof(unsigned int), hipMemcpyHostToDevice);
    manageCudaError();
  }

#elif defined VECTOR_O_64BIT_COMPRESSION

  device->n_count = host->n_count;
  device->m_count = host->m_count;

  for (size_t i=0; i<device->n_count; i++) {
    hipMalloc((void**) &device->count[i], device->m_count * sizeof(unsigned long long));
    manageCudaError();
    hipMemcpy(device->count[i], host->count[i], host->m_count * sizeof(unsigned long long), hipMemcpyHostToDevice);
    manageCudaError();
  }

#endif
}

void freeCompMatrixGPUHost(comp_matrix *matrix) {

for (size_t i=0; i<matrix->n_desp; i++) {
    hipHostFree(matrix->desp[i]);
#if defined VECTOR_O_32BIT_COMPRESSION || VECTOR_O_64BIT_COMPRESSION
    hipHostFree(matrix->count[i]);
#endif
  }

}

void freeCompMatrixGPUDevice(comp_matrix *matrix) {

  for (size_t i=0; i<matrix->n_desp; i++) {
    hipFree(matrix->desp[i]);
#if defined VECTOR_O_32BIT_COMPRESSION || VECTOR_O_64BIT_COMPRESSION
    hipFree(matrix->count[i]);
#endif
  }

}

__global__ void BWExactSearchBackwardGPU(char *W, unsigned int *nW, size_t *k, size_t *l, size_t k_ini, size_t l_ini, unsigned int *C, unsigned int *C1, comp_matrix O) {

  int i;
  BWiterationVariablesGPU();
  size_t k2, l2;
  unsigned int offset  = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ unsigned int Cshared[4];
  __shared__ unsigned int C1shared[4];
  
  if (threadIdx.x<4) {
    Cshared[threadIdx.x]  = C[threadIdx.x];
    C1shared[threadIdx.x] = C1[threadIdx.x];
  }

  __syncthreads();

  k2 = k_ini; l2 = l_ini;

  for (i=nW[offset]-1; (k2<=l2) && (i>=0); i--)
    BWiterationGPU(k2, l2, k2, l2, W[offset*MAXLINE+i], Cshared, C1shared, O);

  k[offset] = k2;
  l[offset] = l2;

}

__global__ void BWExactSearchForwardGPU(char *W, unsigned int *nW, size_t *k, size_t *l, size_t k_ini, size_t l_ini, unsigned int *C, unsigned int *C1, comp_matrix O) {

  int i;
  BWiterationVariablesGPU();
  size_t k2, l2;
  unsigned int offset  = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ unsigned int Cshared[4];
  __shared__ unsigned int C1shared[4];

  if (threadIdx.x<4) {
    Cshared[threadIdx.x] = C[threadIdx.x];
    C1shared[threadIdx.x] = C1[threadIdx.x];
  }

  __syncthreads();

  k2 = k_ini;  l2 = l_ini;

  for (i=0; (k2<=l2) && (i<nW[offset]); i++)
    BWiterationGPU(k2, l2, k2, l2, W[offset*MAXLINE+i], Cshared, C1shared, O);

  k[offset] = k2;
  l[offset] = l2;

}

//-----------------------------------------------------------------------------
// _ex functions, for extended fucntions, 
//
// These functions adds a new parameter num_reads, in addition, the nW vector
// contains indices to the start of each read in W vector, instead of the length
// of the read as the original functions ("no _ex") do
//
//-----------------------------------------------------------------------------

__global__ void BWExactSearchBackwardGPU_ex(char *W, unsigned int *nW, size_t *k, size_t *l, size_t k_ini, size_t l_ini, unsigned int *C, unsigned int *C1, comp_matrix O, size_t num_reads) {

  int i, len;
  BWiterationVariablesGPU();
  size_t k2, l2, index;
  unsigned int offset  = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ unsigned int Cshared[4];
  __shared__ unsigned int C1shared[4];
  
  if (threadIdx.x<4) {
    Cshared[threadIdx.x]  = C[threadIdx.x];
    C1shared[threadIdx.x] = C1[threadIdx.x];
  }

  __syncthreads();

  if (offset < num_reads) {

    k2 = k_ini; l2 = l_ini;
    index = nW[offset];
    len = nW[offset + 1] - index - 1;

    for (i = len - 1; (k2 <= l2) && (i >= 0); i--) {
      BWiterationGPU(k2, l2, k2, l2, W[index + i], Cshared, C1shared, O);
    }
    k[offset] = k2; //k_ini; //k2;
    l[offset] = l2; //l_ini; //l2;
  }
}

//-----------------------------------------------------------------------------

__global__ void BWExactSearchForwardGPU_ex(char *W, unsigned int *nW, size_t *k, size_t *l, size_t k_ini, size_t l_ini, unsigned int *C, unsigned int *C1, comp_matrix O, size_t num_reads) {

  int i, len;
  BWiterationVariablesGPU();
  size_t k2, l2, index;
  unsigned int offset  = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ unsigned int Cshared[4];
  __shared__ unsigned int C1shared[4];

  if (threadIdx.x<4) {
    Cshared[threadIdx.x] = C[threadIdx.x];
    C1shared[threadIdx.x] = C1[threadIdx.x];
  }

  __syncthreads();

  if (offset < num_reads) {
    k2 = k_ini;  l2 = l_ini;
    index = nW[offset];
    len = nW[offset + 1] - index - 1;

    for (i = 0; (k2 <= l2) && (i < len); i++) {
      BWiterationGPU(k2, l2, k2, l2, W[index + i], Cshared, C1shared, O);
    }

    k[offset] = k2; //index; // k_ini; //k2;
    l[offset] = l2; //len; // l_ini; //l2;
  }
}

//-----------------------------------------------------------------------------
// end of _ex functions
//-----------------------------------------------------------------------------


//IN PROGRESS:
__global__ void BWExactFinalResultsBackwardGPU(char *W, unsigned int *C, unsigned int *C1, comp_matrix O, results_list rl_prev, results_list rl_next, unsigned int chunk_size, int *stack_size) {

  BWiterationVariablesGPU();
  size_t k, l;
  int start, pos, pos_start, end;
  unsigned read_index, read_offset;
  unsigned int offset  = blockIdx.x * blockDim.x + threadIdx.x;
  result *r_prev, *r_next;

  __shared__ unsigned int Cshared[4];
  __shared__ unsigned int C1shared[4];

  if (offset==0)
    *stack_size=0;

  if (threadIdx.x<4) {
    Cshared[threadIdx.x] = C[threadIdx.x];
    C1shared[threadIdx.x] = C1[threadIdx.x];
  }

  __syncthreads();

  r_prev = &rl_prev.list[offset];

  start      = r_prev->start;
  pos        = r_prev->pos;
  end        = r_prev->end;
  k          = r_prev->k;
  l          = r_prev->l;
  read_index = r_prev->read_index;
  read_offset = /*read_index*/offset*MAXLINE;

  pos_start = pos - chunk_size;
  if (pos_start < start) pos_start = start;

  for(; pos>=pos_start; pos--) {
    BWiterationGPU(k, l, k, l, (size_t)W[read_offset + pos], Cshared, C1shared, O);
    if (k > l) {
      pos=start-1; break;
    }
  }

  r_next = &rl_next.list[/*atomicAdd(stack_size,1)*/offset];

  r_next->start = start;
  r_next->pos = pos;
  r_next->end = end;
  r_next->k = k;
  r_next->l = l;
  r_next->read_index = read_index;

}

__global__ void BWExactSearchBackwardVectorGPU(char *W, unsigned int *nW, size_t *k, size_t *l, size_t k_ini, size_t l_ini, unsigned int *C, unsigned int *C1, comp_matrix O) {

  int i;
  BWiterationVariablesGPU();
  size_t k2, l2;
  unsigned int offset  = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ unsigned int Cshared[4];
  __shared__ unsigned int C1shared[4];

  if (threadIdx.x<4) {
    Cshared[threadIdx.x] = C[threadIdx.x];
    C1shared[threadIdx.x] = C1[threadIdx.x];
  }

  __syncthreads();

  k2 = k_ini;  l2 = l_ini;

  for (i=nW[offset]-1; i>=0; i--) {

    BWiterationGPU(k2, l2, k2, l2, W[offset*MAXLINE + i], Cshared, C1shared, O);

    k[offset*MAXLINE+i] = k2;
    l[offset*MAXLINE+i] = l2;

    if (k2 > l2) {
      i--;
      break;
    }

  }

  for(;i>=0; i--) {
    k[offset*MAXLINE+i] = k2;
    l[offset*MAXLINE+i] = l2;
  }

}

__global__ void BWExactSearchForwardVectorGPU(char *W, unsigned int *nW, size_t *k, size_t *l, size_t k_ini, size_t l_ini, unsigned int *C, unsigned int *C1, comp_matrix O) {

  int i;
  BWiterationVariablesGPU();
  size_t k2, l2;
  unsigned int offset  = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ unsigned int Cshared[4];
  __shared__ unsigned int C1shared[4];

  if (threadIdx.x<4) {
    Cshared[threadIdx.x] = C[threadIdx.x];
    C1shared[threadIdx.x] = C1[threadIdx.x];
  }

  __syncthreads();

  k2 = k_ini;  l2 = l_ini;

  for (i=0; i<nW[offset]; i++) {

    BWiterationGPU(k2, l2, k2, l2, W[offset*MAXLINE + i], Cshared, C1shared, O);

    k[offset*MAXLINE+i] = k2;
    l[offset*MAXLINE+i] = l2;

  }

}

/*
__global__ void BWExactIterativeSearchGPU(char *W, int *nW, int *nWe, int *k, int *l, int k_ini, int l_ini, int *C,  int *O,  int sizO) {

  int i, b; //, pos;
  unsigned long int k2, l2;
  char val1, val2, val3, val4;
  int siz1, siz2, siz3, siz4;

  int offset  = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ int Cshared[4];

  if (threadIdx.x<4) { // Minimum data is a 32 block
    Cshared[threadIdx.x] = C[threadIdx.x];
  }

  __syncthreads();

  k2 = k_ini;
  l2 = l_ini;

  // First block of 4 bases should not be fully filled

  int real_size = nWe[offset];
  int resto = real_size % 4;
 
  b = W[offset*MAXLINECOMP + nW[offset] - 1];

  switch(resto) {

  case 0:
    val4 = ( b >> 6 ) & 3;
    siz4 = val4*sizO;
    
    k2 = Cshared[val4] + O[siz4 + k2    ] + 1;
    l2 = Cshared[val4] + O[siz4 + l2 + 1];
    
  case 3:
    val3 = ( b >> 4 ) & 3;
    siz3 = val3*sizO;

    k2 = Cshared[val3] + O[siz3 + k2    ] + 1;
    l2 = Cshared[val3] + O[siz3 + l2 + 1];
    
  case 2:
    val2 = ( b >> 2 ) & 3;
    siz2 = val2*sizO;

    k2 = Cshared[val2] + O[siz2 + k2    ] + 1;
    l2 = Cshared[val2] + O[siz2 + l2 + 1];

  case 1:
    val1 = ( b      ) & 3;
    siz1 = val1*sizO;

    k2 = Cshared[val1] + O[siz1 + k2    ] + 1;
    l2 = Cshared[val1] + O[siz1 + l2 + 1];

  }

  __syncthreads();

  for (i=nW[offset]-2; (k2<=l2) && (i>=0); i--) {

    b = W[offset*MAXLINECOMP + i];

    val4 = ( b >> 6 ) & 3;
    siz4 = val4*sizO;

    k2 = Cshared[val4] + O[siz4 + k2    ] + 1;
    l2 = Cshared[val4] + O[siz4 + l2 + 1];

    val3 = ( b >> 4 ) & 3;
    siz3 = val3*sizO;

    k2 = Cshared[val3] + O[siz3 + k2    ] + 1;
    l2 = Cshared[val3] + O[siz3 + l2 + 1];
    
    val2 = ( b >> 2 ) & 3;
    siz2 = val2*sizO;

    k2 = Cshared[val2] + O[siz2 + k2    ] + 1;
    l2 = Cshared[val2] + O[siz2 + l2 + 1];

    val1 = ( b      ) & 3;
    siz1 = val1*sizO;

    k2 = Cshared[val1] + O[siz1 + k2    ] + 1;
    l2 = Cshared[val1] + O[siz1 + l2 + 1];

  }

  __syncthreads();

   k[offset] = k2;
   l[offset] = l2;

}

__global__ void BWExactIterativeSearchGPURev(char *W, int *nW, int *nWe, int *k, int *l, int k_ini, int l_ini, int *C,  int *O,  int sizO) {

  int i, b;//, pos;
  unsigned long int k2, l2;
  char val1, val2, val3, val4;
  int siz1, siz2, siz3, siz4;

  int offset  = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ int Cshared[4];

  if (threadIdx.x<4) { // Minimum data is a 32 block
    Cshared[threadIdx.x] = C[threadIdx.x];
  }
 
  __syncthreads();

  k2 = k_ini;
  l2 = l_ini;

  for (i=0; (k2<=l2) && (i<nW[offset]-1); i++) {

    b = W[offset*MAXLINECOMP + i];

    val1 = ( b      ) & 3;
    siz1 = val1*sizO;

    k2 = Cshared[val1] + O[siz1 + k2    ] + 1;
    l2 = Cshared[val1] + O[siz1 + l2 + 1];

    val2 = ( b >> 2 ) & 3;
    siz2 = val2*sizO;

    k2 = Cshared[val2] + O[siz2 + k2    ] + 1;
    l2 = Cshared[val2] + O[siz2 + l2 + 1];
    
    val3 = ( b >> 4 ) & 3;
    siz3 = val3*sizO;

    k2 = Cshared[val3] + O[siz3 + k2    ] + 1;
    l2 = Cshared[val3] + O[siz3 + l2 + 1];

    val4 = ( b >> 6 ) & 3;
    siz4 = val4*sizO;
    
    k2 = Cshared[val4] + O[siz4 + k2    ] + 1;
    l2 = Cshared[val4] + O[siz4 + l2 + 1];

  }

   __syncthreads();

  if (k2<=l2) {

    // Last block of 4 bases should not be fully filled

    int real_size = nWe[offset];
    int resto = real_size % 4;

    b = W[offset*MAXLINECOMP + nW[offset] - 1];

    switch(resto) {

    case 0:

      val1 = ( b      ) & 3;
      siz1 = val1*sizO;

      k2 = Cshared[val1] + O[siz1 + k2    ] + 1;
      l2 = Cshared[val1] + O[siz1 + l2 + 1];

      val2 = ( b >> 2 ) & 3;
      siz2 = val2*sizO;

      k2 = Cshared[val2] + O[siz2 + k2    ] + 1;
      l2 = Cshared[val2] + O[siz2 + l2 + 1];

      val3 = ( b >> 4 ) & 3;
      siz3 = val3*sizO;

      k2 = Cshared[val3] + O[siz3 + k2    ] + 1;
      l2 = Cshared[val3] + O[siz3 + l2 + 1];

      val4 = ( b >> 6 ) & 3;
      siz4 = val4*sizO;
    
      k2 = Cshared[val4] + O[siz4 + k2    ] + 1;
      l2 = Cshared[val4] + O[siz4 + l2 + 1];
  
      break;

    case 3:

      val1 = ( b      ) & 3;
      siz1 = val1*sizO;

      k2 = Cshared[val1] + O[siz1 + k2    ] + 1;
      l2 = Cshared[val1] + O[siz1 + l2 + 1];

      val2 = ( b >> 2 ) & 3;
      siz2 = val2*sizO;

      k2 = Cshared[val2] + O[siz2 + k2    ] + 1;
      l2 = Cshared[val2] + O[siz2 + l2 + 1];

      val3 = ( b >> 4 ) & 3;
      siz3 = val3*sizO;

      k2 = Cshared[val3] + O[siz3 + k2    ] + 1;
      l2 = Cshared[val3] + O[siz3 + l2 + 1];

      break;

    case 2:

      val1 = ( b      ) & 3;
      siz1 = val1*sizO;

      k2 = Cshared[val1] + O[siz1 + k2    ] + 1;
      l2 = Cshared[val1] + O[siz1 + l2 + 1];

      val2 = ( b >> 2 ) & 3;
      siz2 = val2*sizO;

      k2 = Cshared[val2] + O[siz2 + k2    ] + 1;
      l2 = Cshared[val2] + O[siz2 + l2 + 1];

      break;

    case 1:
      val1 = ( b      ) & 3;
      siz1 = val1*sizO;

      k2 = Cshared[val1] + O[siz1 + k2    ] + 1;
      l2 = Cshared[val1] + O[siz1 + l2 + 1];

      break;
    
    }

  }

  __syncthreads();

  k[offset] = k2;
  l[offset] = l2;

}
*/

void BWExactSearchBackwardGPUWrapper_ex(unsigned int num_bloques, unsigned int tam_bloques, char *W, unsigned int *nW, size_t *k, size_t *l, size_t k_ini, size_t l_ini, vector *C, vector *C1, comp_matrix *O, size_t num_reads) {
  BWExactSearchBackwardGPU_ex<<<num_bloques,tam_bloques>>>(W, nW, k, l, k_ini, l_ini, C->vector, C1->vector, *O, num_reads);
}

void BWExactSearchForwardGPUWrapper_ex(unsigned int num_bloques, unsigned int tam_bloques, char *W, unsigned int *nW, size_t *k, size_t *l, size_t k_ini, size_t l_ini, vector *C, vector *C1, comp_matrix *O, size_t num_reads) {
  BWExactSearchForwardGPU_ex<<<num_bloques,tam_bloques>>>(W, nW, k, l, k_ini, l_ini, C->vector, C1->vector, *O, num_reads);
}




void BWExactSearchBackwardGPUWrapper(unsigned int num_bloques, unsigned int tam_bloques, char *W, unsigned int *nW, size_t *k, size_t *l, size_t k_ini, size_t l_ini, vector *C, vector *C1, comp_matrix *O) {
  BWExactSearchBackwardGPU<<<num_bloques,tam_bloques>>>(W, nW, k, l, k_ini, l_ini, C->vector, C1->vector, *O);
}

void BWExactSearchForwardGPUWrapper(unsigned int num_bloques, unsigned int tam_bloques, char *W, unsigned int *nW, size_t *k, size_t *l, size_t k_ini, size_t l_ini, vector *C, vector *C1, comp_matrix *O) {
  BWExactSearchForwardGPU<<<num_bloques,tam_bloques>>>(W, nW, k, l, k_ini, l_ini, C->vector, C1->vector, *O);
}

void BWExactFinalResultsBackwardGPUWrapper(unsigned int num_bloques, unsigned int tam_bloques, char *W, vector *C, vector *C1, comp_matrix *O, results_list *rl_prev, results_list *rl_next, unsigned int chunk_size, int *stack_size) {
  BWExactFinalResultsBackwardGPU<<<num_bloques,tam_bloques>>>(W, C->vector, C1->vector, *O, *rl_prev, *rl_next, chunk_size, stack_size);
}

void BWExactSearchBackwardVectorGPUWrapper(unsigned int num_bloques, unsigned int tam_bloques, char *W, unsigned int *nW, size_t *k, size_t *l, size_t k_ini, size_t l_ini, vector *C, vector *C1, comp_matrix *O) {
  BWExactSearchBackwardVectorGPU<<<num_bloques,tam_bloques>>>(W, nW, k, l, k_ini, l_ini, C->vector, C1->vector, *O);
}

void BWExactSearchForwardVectorGPUWrapper(unsigned int num_bloques, unsigned int tam_bloques, char *W, unsigned int *nW, size_t *k, size_t *l, size_t k_ini, size_t l_ini, vector *C, vector *C1, comp_matrix *O) {
  BWExactSearchForwardVectorGPU<<<num_bloques,tam_bloques>>>(W, nW, k, l, k_ini, l_ini, C->vector, C1->vector, *O);
}
